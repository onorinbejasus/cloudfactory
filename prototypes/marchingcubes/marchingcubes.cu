#include "hip/hip_runtime.h"
#include "marchingcubes.hh"

list<vertex> vertexList;

__device__ __host__
int getIndex(int x, int y, int z, int X, int Y, int Z){
	
	return x + (y * X) + (z * X * Y);
}

__global__
void drawMetaball(float *voxels, int sizeX, int sizeY, int sizeZ, int px, int py, int pz, int R){
	
 	// calculate the unique thread index
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	float x = index%sizeX;
 	float y = index/sizeY;
	
	for(int z = 0; z < sizeZ; z++)		
		voxels[ (int)(x + (y*sizeX) + (z * sizeX * sizeY)) ] += ( R / (float)( (x-px)*(x-px) + (y-py)*(y-py) + (z-pz)*(z-pz)*4 ) ) ;
		
		//voxels[x][y][z] += ( (R) / (float)( (x-px)*(x-px) + (y-py)*(y-py) + (z-pz)*(z-pz)*4 ) );
		
					
}

vertex interpolate(double isolevel, vertex p1, vertex p2, float valp1, float valp2) {
    if(fabs(isolevel - valp1) < 0.00001)
        return p1;
    if(fabs(isolevel - valp2) < 0.00001)
        return p2;
    if(fabs(valp1 - valp2) < 0.00001)
        return p1;

    vertex p;
    double diff = (double)(isolevel - valp1) / (valp2 - valp1);
    p.x = p1.x + diff * (p2.x - p1.x);
    p.y = p1.y + diff * (p2.y - p1.y);
    p.z = p1.z + diff * (p2.z - p1.z);

    p.normal_x = p1.normal_x + diff * (p2.normal_x - p1.normal_x);
    p.normal_y = p1.normal_y + diff * (p2.normal_y - p1.normal_y);
    p.normal_z = p1.normal_z + diff * (p2.normal_z - p1.normal_z);

    return p;
}

void processCube(cube cube, double isolevel) {
    int cubeindex = 0;
    if(cube.val[0] > isolevel) cubeindex |= 1;
    if(cube.val[1] > isolevel) cubeindex |= 2;
    if(cube.val[2] > isolevel) cubeindex |= 4;
    if(cube.val[3] > isolevel) cubeindex |= 8;
    if(cube.val[4] > isolevel) cubeindex |= 16;
    if(cube.val[5] > isolevel) cubeindex |= 32;
    if(cube.val[6] > isolevel) cubeindex |= 64;
    if(cube.val[7] > isolevel) cubeindex |= 128;

    // Cube is entirely in/out of the surface
    if(edgeTable[cubeindex] == 0 || edgeTable[cubeindex] == 255)
        return;

    vertex vertlist[12];
    // Find the vertices where the surface intersects the cube
    if(edgeTable[cubeindex] & 1)
        vertlist[0] = interpolate(isolevel,cube.p[0],cube.p[1],cube.val[0],cube.val[1]);
    if(edgeTable[cubeindex] & 2)
        vertlist[1] = interpolate(isolevel,cube.p[1],cube.p[2],cube.val[1],cube.val[2]);
    if(edgeTable[cubeindex] & 4)
        vertlist[2] = interpolate(isolevel,cube.p[2],cube.p[3],cube.val[2],cube.val[3]);
    if(edgeTable[cubeindex] & 8)
        vertlist[3] = interpolate(isolevel,cube.p[3],cube.p[0],cube.val[3],cube.val[0]);
    if(edgeTable[cubeindex] & 16)
        vertlist[4] = interpolate(isolevel,cube.p[4],cube.p[5],cube.val[4],cube.val[5]);
    if(edgeTable[cubeindex] & 32)
        vertlist[5] = interpolate(isolevel,cube.p[5],cube.p[6],cube.val[5],cube.val[6]);
    if(edgeTable[cubeindex] & 64)
        vertlist[6] = interpolate(isolevel,cube.p[6],cube.p[7],cube.val[6],cube.val[7]);
    if(edgeTable[cubeindex] & 128)
        vertlist[7] = interpolate(isolevel,cube.p[7],cube.p[4],cube.val[7],cube.val[4]);
    if(edgeTable[cubeindex] & 256)
        vertlist[8] = interpolate(isolevel,cube.p[0],cube.p[4],cube.val[0],cube.val[4]);
    if(edgeTable[cubeindex] & 512)
        vertlist[9] = interpolate(isolevel,cube.p[1],cube.p[5],cube.val[1],cube.val[5]);
    if(edgeTable[cubeindex] & 1024)
        vertlist[10] = interpolate(isolevel,cube.p[2],cube.p[6],cube.val[2],cube.val[6]);
    if(edgeTable[cubeindex] & 2048)
        vertlist[11] = interpolate(isolevel,cube.p[3],cube.p[7],cube.val[3],cube.val[7]);

    for(int i = 0; triTable[cubeindex][i] != -1; i++) {
        vertexList.push_back(vertlist[triTable[cubeindex][i]]);
    }
}

void clearVertexList(){
	vertexList.erase(vertexList.begin(),vertexList.end());
}

list<vertex>& runMarchingCubes(float *voxels, int sizeX, int sizeY, int sizeZ, 
        						int stepX, int stepY, int stepZ, double isovalue) {
    // Run the processCube function on every cube in the grid
	for(int x = stepX; x < sizeX-2*stepX; x += stepX) {
		for(int y = stepY; y < sizeY-2*stepY; y += stepY) {
			for(int z = stepZ; z < sizeZ-2*stepZ; z += stepZ) {
					
				cube c = {{
					{x,y,z, 
                        (double)(voxels[getIndex(x+ stepX, y, z, sizeX, sizeY, sizeZ)]- voxels[getIndex(x - stepX, y, z, sizeX, sizeY, sizeZ)]) / -stepX,
                        (double)(voxels[getIndex(x, y+stepY, z, sizeX, sizeY, sizeZ)]-voxels[getIndex(x, y-stepY, z, sizeX, sizeY, sizeZ)]) / -stepY,
                        (double)(voxels[getIndex(x, y, z + stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x, y, z-stepZ, sizeX, sizeY, sizeZ)]) / -stepZ
                    },
					{x+stepX,y,z, 
                        (double)(voxels[getIndex(x+2*stepX,y,z, sizeX, sizeY, sizeZ)]-voxels[getIndex(x,y,z, sizeX, sizeY, sizeZ)]) / -stepX,
                        (double)(voxels[getIndex(x+stepX,y+stepY,z, sizeX, sizeY, sizeZ)]-voxels[getIndex(x+stepX,y-stepY,z, sizeX, sizeY, sizeZ)]) / -stepY,
                        (double)(voxels[getIndex(x+stepX,y,z+stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x+stepX,y,z-stepZ, sizeX, sizeY, sizeZ)]) / -stepZ
                    },
					{x+stepX,y,z+stepZ, 
                        (double)(voxels[getIndex(x+2*stepX,y,z+stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x,y,z+stepZ, sizeX, sizeY, sizeZ)]) / -stepX,
                        (double)(voxels[getIndex(x+stepX,y+stepY,z+stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x+stepX,y-stepY,z+stepZ, sizeX, sizeY, sizeZ)]) / -stepY,
                        (double)(voxels[getIndex(x+stepX,y,z+2*stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x+stepX,y,z, sizeX, sizeY, sizeZ)]) / -stepZ
                    },
					{x,y,z+stepZ, 
                        (double)(voxels[getIndex(x+stepX,y,z+stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x-stepX,y,z+stepZ, sizeX, sizeY, sizeZ)]) / -stepX,
                        (double)(voxels[getIndex(x,y+stepY,z+stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x,y-stepY,z+stepZ, sizeX, sizeY, sizeZ)]) / -stepY,
                        (double)(voxels[getIndex(x,y,z+2*stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x,y,z, sizeX, sizeY, sizeZ)]) / -stepZ
                    },
					{x,y+stepY,z, 
                        (double)(voxels[getIndex(x+stepX,y+stepY,z, sizeX, sizeY, sizeZ)]-voxels[getIndex(x-stepX,y+stepY,z, sizeX, sizeY, sizeZ)]) / -stepX,
                        (double)(voxels[getIndex(x,y+2*stepY,z, sizeX, sizeY, sizeZ)]-voxels[getIndex(x,y,z, sizeX, sizeY, sizeZ)]) / -stepY,
                        (double)(voxels[getIndex(x,y+stepY,z+stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x,y+stepY,z-stepZ, sizeX, sizeY, sizeZ)]) / -stepZ
                    },
					{x+stepX,y+stepY,z, 
                        (double)(voxels[getIndex(x+2*stepX,y+stepY,z, sizeX, sizeY, sizeZ)]-voxels[getIndex(x+stepX,y+stepY,z, sizeX, sizeY, sizeZ)]) / -stepX,
                        (double)(voxels[getIndex(x+stepX,y+2*stepY,z, sizeX, sizeY, sizeZ)]-voxels[getIndex(x+stepX,y,z, sizeX, sizeY, sizeZ)]) / -stepY,
                        (double)(voxels[getIndex(x+stepX,y+stepY,z+stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x+stepX,y+stepY,z-stepZ, sizeX, sizeY, sizeZ)]) / -stepZ
                    },
					{x+stepX,y+stepY,z+stepZ, 
                        (double)(voxels[getIndex(x+2*stepX,y+stepY,z+stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x,y+stepY,z+stepZ, sizeX, sizeY, sizeZ)]) / -stepX,
                        (double)(voxels[getIndex(x+stepX,y+2*stepY,z+stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x+stepX,y,z+stepZ, sizeX, sizeY, sizeZ)]) / -stepY,
                        (double)(voxels[getIndex(x+stepX,y+stepY,z+2*stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x+stepX,y+stepY,z, sizeX, sizeY, sizeZ)]) / -stepZ
                    },
					{x,y+stepY,z+stepZ, 
                        (double)(voxels[getIndex(x+stepX,y+stepY,z+stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x-stepX,y+stepY,z+stepZ, sizeX, sizeY, sizeZ)]) / -stepX,
                        (double)(voxels[getIndex(x,y+2*stepY,z+stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x,y,z+stepZ, sizeX, sizeY, sizeZ)]) / -stepY,
                        (double)(voxels[getIndex(x,y+stepY,z+2*stepZ, sizeX, sizeY, sizeZ)]-voxels[getIndex(x,y+stepY,z, sizeX, sizeY, sizeZ)]) / -stepZ
                    }
				},{
					voxels[getIndex(x,y,z, sizeX, sizeY, sizeZ)],
					voxels[getIndex(x+stepX,y,z, sizeX, sizeY, sizeZ)],
					voxels[getIndex(x+stepX,y,z+stepZ, sizeX, sizeY, sizeZ)],
					voxels[getIndex(x,y,z+stepZ, sizeX, sizeY, sizeZ)],
					voxels[getIndex(x,y+stepY,z, sizeX, sizeY, sizeZ)],
					voxels[getIndex(x+stepX,y+stepY,z, sizeX, sizeY, sizeZ)],
					voxels[getIndex(x+stepX,y+stepY,z+stepZ, sizeX, sizeY, sizeZ)],
					voxels[getIndex(x,y+stepY,z+stepZ, sizeX, sizeY, sizeZ)]
				}};
				processCube(c, isovalue);
			}
		}
	}

	return vertexList;
}
